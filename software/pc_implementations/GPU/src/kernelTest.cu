/*    Sample program to test custom kernel for
      Matrix multiplication algorithm
      adapted from the youtube channel CoffeeBeforeArch

      compile with:
         > make cuda
*/


#include <cstdlib>
#include <cassert>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


#include <hipblas.h>


using namespace std;

#define BLOCK (512)

void error(const char *s)
{
  cout << s << endl;
  assert(0);
  exit(-1);
}

dim3 cuda_gridsize(size_t n){
    size_t k = (n-1) / BLOCK + 1;
    size_t x = k;
    size_t y = 1;
    if(x > 65535){
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    //printf("%ld %ld %ld %ld\n", n, x, y, x*y*BLOCK);
    return d;
}


void check_error(hipError_t status)
{
    //cudaDeviceSynchronize();
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        cout << "CUDA Error:" << s << endl;
        assert(0);
    } 
    if (status2 != hipSuccess)
    {   
        const char *s = hipGetErrorString(status);
        cout << "CUDA Error Prev: " <<s << endl;
        assert(0);
    } 
}

__global__ void fill_kernel(int N, int ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

void fill_gpu(int N, int ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}


float *cuda_make_array(float *x, size_t n)
{
  float *x_gpu;
  size_t size = sizeof(float)*n;
  hipError_t status = hipMalloc((void**)&x_gpu, size);
  check_error(status);
  if(x){
    status = hipMemcpy(x_gpu, x, size, hipMemcpyHostToDevice);
    check_error(status);
  } else {
    fill_gpu(n, 0, x_gpu, 1);
  }
  if(!x_gpu) error("Cuda malloc failed\n");
  return x_gpu;
}

void cuda_free(void *x_gpu)
{
    hipError_t status = hipFree(x_gpu);
    check_error(status);
}

void cuda_pull_array(float *x_gpu, float* x, size_t n)
{
  size_t size = sizeof(float)*n;
  hipError_t status = hipMemcpy(x, x_gpu, size, hipMemcpyDeviceToHost);
  check_error(status);
}


void cuda_pull_fp_array(int *x_gpu, int* x, size_t n){
  size_t size = sizeof(int)*n;
  hipError_t status = hipMemcpy(x, x_gpu, size, hipMemcpyDeviceToHost);
  check_error(status);
}


__global__ void copy_fp2float_array_kernel(int N, int *X, float *Y, int fracFP)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if(i < N) Y[i] = (float) ((X[i]*1.0)/((int)(1<<fracFP)));
}

void copy_fp2float_array_gpu(int N, int *X, float* Y, int fracFP)
{
  copy_fp2float_array_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, Y, fracFP);
  check_error(hipPeekAtLastError());
}


__global__ void copy_float2fp_array_kernel(int N, float *X, int *Y, int fracFP)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if(i < N) Y[i] = (int) (X[i]*((int)(1<<fracFP)));
}

void copy_float2fp_array_gpu(int N, float *X, int* Y, int fracFP)
{
  copy_float2fp_array_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, Y, fracFP);
  check_error(hipPeekAtLastError());
}


//////////////////////////////////

// Naive Matrix Multiplication implementation for square matrices
__global__ void matrixMul(float *a, float *b, float *c, int N){
	// get thread global id
	int row  = blockIdx.x * blockDim.x + threadIdx.x;
	int col  = blockIdx.y * blockDim.y + threadIdx.y;

	// check range
	if(row < N && col < N){

		// Accumulate partial result
		float tmp = 0;

		for (int i = 0; i < N; i++){
			tmp += a[row*N + i]*b[i*N + col];	
		}

		//Write back the result
		c[row*N + col] = tmp;
	}
}


// Naive Matrix Multiplication implementation for rectangular matrices
__global__ void matrixMulRect(float *a, float *b, float *c, int M, int N, int K){
	// get thread global id
	int row  = blockIdx.x * blockDim.x + threadIdx.x;
	int col  = blockIdx.y * blockDim.y + threadIdx.y;

	// check range
	if(row < M && col < N){

		// Accumulate partial result
		float tmp = 0;

		for (int i = 0; i < K; i++){
			tmp += a[row*K + i]*b[i*N + col];	
		}

		//Write back the result
		c[row*N + col] = tmp;
	}
}

// Naive Matrix Multiplication implementation for rectangular matrices
__global__ void matrixMulRect_fp(int16_t *a, int16_t *b, int16_t *c, int M, int N, int K){
	// get thread global id
	int row  = blockIdx.x * blockDim.x + threadIdx.x;
	int col  = blockIdx.y * blockDim.y + threadIdx.y;

	// check range
	if(row < M && col < N){

		// Accumulate partial result
		int tmp = 0;

		for (int i = 0; i < K; i++){
		  tmp += ((int) a[row*K + i])*((int)b[i*N + col]);	
		}

		//Write back the result
		c[row*N + col] = (int16_t) (tmp >> 10);
	}
}


////////////////////////////////

//Shared version
// statically define shared memory
#define SHMEM_SIZE (16*16)

__global__ void matrixMulShared(float *a, float *b, float *c, int m, int n, int k){
	// shared memory
	__shared__ float A[SHMEM_SIZE];
	__shared__ float B[SHMEM_SIZE];

	// get global thread id
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// Extract some builtin values to simplify code
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dim = blockDim.x;

	//number of blocks along k dimention
	int num_k_blocks = (k + dim - 1)/dim;
	int idx_a, idx_b;
	
	// move the tile across the length of the grid
	float tmp = 0;
	// accumulate along k blocks
	for (int i = 0; i < num_k_blocks; i++){
	  // load values into shared memory
	  // Each thread in a block loads 1 value for each A and B
	  
	  // calculate matrix indexes
	  idx_a = (row*k) + (i*dim) + ty;
	  idx_b = (i*dim*n) + (tx*n) + col;

	  // check input matrices boundaries
	  A[tx * dim + ty] = (idx_a < m*k) ? a[idx_a] : 0;
	  B[tx * dim + ty] = (idx_b < k*n) ? b[idx_b] : 0;

	  __syncthreads();

	  //only perform muladd inside output boundaries
	  if(row < m && col < n){
	    //Accumulate the partial results
	    for (int j = 0; j < dim; j++){
	      tmp += A[tx*dim + j] * B[j*dim + ty];
	    }
	  }
	  __syncthreads();
	}

	// write back result to main memory
	c[row*n + col] = tmp;
}


//Shared version
// statically define shared memory
#define SHMEM_SIZE (16*16)

__global__ void matrixMul_fp_Shared(int *a, int *b, int *c, int m, int n, int k){
	// shared memory
	__shared__ int A[SHMEM_SIZE];
	__shared__ int B[SHMEM_SIZE];

	// get global thread id
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// Extract some builtin values to simplify code
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dim = blockDim.x;

	//number of blocks along k dimention
	int num_k_blocks = (k + dim - 1)/dim;
	int idx_a, idx_b;

	int tmp = 0;
	// accumulate along k blocks
	for (int i = 0; i < num_k_blocks; i++){
	  // load values into shared memory
	  // Each thread in a block loads 1 value for each A and B
	  
	  // calculate matrix indexes
	  idx_a = (row*k) + (i*dim) + ty;
	  idx_b = (i*dim*n) + (tx*n) + col;

	  // check index boundaries
	  A[tx * dim + ty] = (idx_a < m*k) ? a[idx_a] : 0;
	  B[tx * dim + ty] = (idx_b < k*n) ? b[idx_b] : 0;

	  __syncthreads();

	  //only perform muladd inside output boundaries
	  if(row < m && col < n){
	    //Accumulate the partial results
	    for (int j = 0; j < dim; j++){
	      tmp += A[tx*dim + j] * B[j*dim + ty];
	    }
	  }
	  __syncthreads();
	}

	// write back result to main memory
	c[row*n + col] = tmp;
}



///////////////////////////


///////////////////////////
// im2col_gpu

//im2col_gpu implementation
// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_gpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}



__global__ void im2col_fp_gpu_kernel(const int n, const int* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        int *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        int* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const int* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_fp_gpu(int *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, int *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_fp_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}



///////////////////////////

///////////////////////////
int cuda_get_device()
{
    int n = 0;
    hipError_t status = hipGetDevice(&n);
    check_error(status);
    return n;
}


hipblasHandle_t blas_handle()
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int i = cuda_get_device();
    if(!init[i]) {
        hipblasCreate(&handle[i]);
        init[i] = 1;
    }
    return handle[i];
}


// gemm_gpu()
void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, 
        float *A_gpu, int lda, 
        float *B_gpu, int ldb,
        float BETA,
        float *C_gpu, int ldc)
{
    hipblasHandle_t handle = blas_handle();
    hipblasStatus_t status = hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N), 
            (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
    //check_error(status);
}



///////////////////////////




//Create CPU side matrix - rows x cols
float* create_matrix(int rows, int cols){
  //create matrix on CPU side
  float* m = (float*) malloc(rows*cols*sizeof(float));
  if(m == 0){
    cout << "Failed to allocate memory" << endl;
    exit(1);
  }
  
  return m;
}


//Create CPU side matrix - rows x cols
int* create_fp_matrix(int rows, int cols){
  //create matrix on CPU side
  int* m = (int*) malloc(rows*cols*sizeof(int));
  if(m == 0){
    cout << "Failed to allocate memory" << endl;
    exit(1);
  }
  
  return m;
}


float* create_3D_array(int w, int h, int c){

  float * array = (float*) malloc(sizeof(float)*w*h*c);
  if(array == 0){
    cout << "Failed to allocate 3D array" << endl;
    exit(1);
  }
  
  return array;
}

int* create_3D_fp_array(int w, int h, int c){

  int * array = (int*) malloc(sizeof(int)*w*h*c);
  if(array == 0){
    cout << "Failed to allocate 3D array" << endl;
    exit(1);
  }
  
  return array;
}


// Initialize matrices - 0 to 100
void init_matrix(int* matrix, int rows, int cols ){

  for (int i = 0; i < rows*cols; i++){
    matrix[i] = i+1; //rand() % 100;
  }

}

void print_matrix(float* matrix, int rows, int cols){
  int i, j, idx=0;
  for(i=0;i<rows;i++){
    cout << "\t";
    for(j=0;j<cols;j++){
      cout << matrix[idx] << " \t";
      idx++;
    }
    cout << endl;
  }

}

void print_fp_matrix(int* matrix, int rows, int cols){
  int i, j, idx=0;
  for(i=0;i<rows;i++){
    cout << "\t";
    for(j=0;j<cols;j++){
      cout << matrix[idx] << " \t";
      idx++;
    }
    cout << endl;
  }

}

// Print 100 evenly spaced positions of the matrix
void print_sample_matrix(float* matrix, int rows, int cols){
  int idx, n_prints=100;
  int step = (rows*cols)/n_prints;

  /*check for smaller matrices < 100 positions*/
  if(rows*cols < n_prints){
    step = 1;
  }

  cout << "\nSampling matrix[" << rows << "][" << cols << "]:" << endl;
  for(idx=0;idx<rows*cols;idx+=step){
    cout << idx << ": " << matrix[idx] << endl;
  }
}

void print_sample_side_by_side(float *a, float* b, int rows, int cols){
  int idx, n_prints=100;
  int step = (rows*cols)/n_prints;

  /*check for smaller matrices < 100 positions*/
  if(rows*cols < n_prints){
    step = 1;
  }

  cout << "\nSampling matrices[" << rows << "][" << cols << "]:" << endl;
  for(idx=0;idx<rows*cols;idx+=step){
    cout << idx << ": " << a[idx] << "\t\t"<< b[idx] << "\tdiff:" << a[idx]-b[idx] << endl;
  }
}


// Initialize 3D array
void init_3D_array(float *im, int w, int h, int c){

  int i, j, k, idx=0;
  for(i=0;i<c;i++){ // channels
    for(j=0;j<h;j++){ //rows
      for(k=0;k<w;k++){ //cols
	im[idx] = rand()%100; //idx+1;
	idx++;
      }
    }
  }
}

// Initialize 3D array
void init_3D_fp_array(int *im, int w, int h, int c){

  int i, j, k, idx=0;
  for(i=0;i<c;i++){ // channels
    for(j=0;j<h;j++){ //rows
      for(k=0;k<w;k++){ //cols
	im[idx] = rand()%100; //idx+1;
	idx++;
      }
    }
  }
}


// Identity kernel
void init_3D_kernel(float *kernel, int w, int h, int c, int num_ker){

  int i, j, k, l, idx=0;
  // int factor = 0;
  // int mid = w/2;
  for(l=0;l<num_ker;l++){ //3D kernel
    // factor = l+1;
    for(i=0;i<c;i++){ // channels
      for(j=0;j<h;j++){ //rows
	for(k=0;k<w;k++){ //cols
	  kernel[idx] = rand()%100;//factor*(j==mid && k==mid);
	  idx++;
	}
      }
    }
  }
}

// Identity kernel
void init_3D_fp_kernel(int *kernel, int w, int h, int c, int num_ker){

  int i, j, k, l, idx=0;
  // int factor = 0;
  // int mid = w/2;
  for(l=0;l<num_ker;l++){ //3D kernel
    // factor = l+1;
    for(i=0;i<c;i++){ // channels
      for(j=0;j<h;j++){ //rows
	for(k=0;k<w;k++){ //cols
	  kernel[idx] = rand()%100;//factor*(j==mid && k==mid);
	  idx++;
	}
      }
    }
  }
}


// Print 3D array
void print_3D_array(float * a, int w, int h, int c){
  
  int i, j, k, idx=0;
  for(i=0;i<c;i++){ // channels
    cout << "c = " << i << endl;
    for(j=0;j<h;j++){ //rows
      cout << "\t";
      for(k=0;k<w;k++){ //cols
	cout << a[idx] << " \t";
	idx++;
      }
      cout << endl;
    }
    cout << "\n" << endl;
  }  
}

// Print 3D array
void print_3D_fp_array(int * a, int w, int h, int c){
  
  int i, j, k, idx=0;
  for(i=0;i<c;i++){ // channels
    cout << "c = " << i << endl;
    for(j=0;j<h;j++){ //rows
      cout << "\t";
      for(k=0;k<w;k++){ //cols
	cout << a[idx] << " \t";
	idx++;
      }
      cout << endl;
    }
    cout << "\n" << endl;
  }  
}


// Verify result of CPU
void verify_result(int *a, int *b, int *c, int c_rows, int c_cols, int a_cols){
  int tmp;
  for (int i = 0; i < c_rows; i++){ // row
    for (int j = 0; j < c_cols; j++){ //col
      // calculate 1 output
      tmp = 0;
      for (int k = 0; k < a_cols; k++){
	tmp += a[i*a_cols + k] * b[k*c_cols + j];
      }

      // Check each result
      assert(tmp == c[i*c_cols + j]);

      // cout << "C[" << i <<"][" << j <<"]:" << tmp << endl; 
      // cout << "C_gpu[" << i <<"][" << j <<"]:" << c[i*c_cols + j] << endl;
    }
  }

}

int main(){
  // input image 4x4
  int w=416, h=416, c=3;
  int ksize=3, num_ker=16;

  
  // convolution input (3D: C x H x W)

  float *im = create_3D_array(w, h, c);

  init_3D_array(im, w, h, c);

  // cout << "Image in:" << endl;
  // print_3D_array(im, w, h, c);

  // im2col unrolled image: (Ksize x Ksize x C) x (H x W)
  float *out = create_matrix(ksize*ksize*c, w*h);


  // weights (Num_ker) x (Ksize x Ksize x C)
  float *weights = create_matrix(num_ker, ksize*ksize*c);
  init_3D_kernel(weights, ksize, ksize, c, num_ker);

  // cout << "Weights:" << endl;

  // print_3D_array(weights, ksize, ksize, c);
  // print_matrix(weights, num_ker, ksize*ksize*c);

  // gemm output (Num_ker x H x W)
  float *gemm_out = create_3D_array(w, h, num_ker);

  //GPU side
  float *im_gpu, *out_gpu;
  im_gpu = cuda_make_array(im, w*h*c);
  out_gpu = cuda_make_array(0, w*h*ksize*ksize*c);
  float* weights_gpu = cuda_make_array(weights, ksize*ksize*c*num_ker);
  float* gemm_out_gpu = cuda_make_array(0, w*h*num_ker);

  im2col_gpu(im_gpu, c, h, w, ksize, 1, 1, out_gpu);

  cuda_pull_array(out_gpu, out, w*h*ksize*ksize*c);

  // cout << "im2col output" << endl;

  // print_sample_matrix(out, ksize*ksize*c, w*h);

  // print_matrix(out, ksize*ksize*c, w*h);

  //gemm
  /* gemm_gpu(0,0,m,n,k,1,a,k,b,n,0,out,n); */
  gemm_gpu(0,0,num_ker,w*h,ksize*ksize*c, 1, weights_gpu, ksize*ksize*c, out_gpu, w*h, 0, gemm_out_gpu, w*h );
  
  cuda_pull_array(gemm_out_gpu, gemm_out, w*h*num_ker);


  // cout << "gemm output:" << endl;
  // print_3D_array(gemm_out, w, h, num_ker);
  // print_sample_matrix(gemm_out, num_ker, w*h);

  // custom kernel
  float * mm_out = create_matrix(num_ker, w*h);
  float * mm_out_gpu = cuda_make_array(0,num_ker*w*h);

  // M, N, K variables
  int M = num_ker;
  int N = w*h;
  int K = ksize*ksize*c;

  //Create CTA and Grid dimentions
  int threads = 16; // 2D => 16 + 16 = 256
  int blocks_row = (M + threads - 1) / threads;
  int blocks_col = (N + threads - 1) / threads;

  // Setup our kernel launch parameters
  dim3 THREADS(threads, threads);
  dim3 BLOCKS(blocks_row, blocks_col);

  clock_t start, end;
  double kernel_time = 0;


  //start time count
  start = clock();

  // Launch kernel
  // Naive implementation
  // matrixMulRect<<<BLOCKS, THREADS, 0, 0>>>(weights_gpu, out_gpu, mm_out_gpu, M, N, K);
  // cudaDeviceSynchronize();

  // Launch kernel
  // Shared memory kernel implementation
  matrixMulShared<<<BLOCKS, THREADS, 0, 0>>>(weights_gpu, out_gpu, mm_out_gpu, M, N, K);
  hipDeviceSynchronize();


  //end time count
  end = clock();
  kernel_time = ((double) (end-start)) / CLOCKS_PER_SEC;
  cout.precision(2);
  cout << "KERNEL TIME: " << fixed << kernel_time << "seconds" << endl;

  //copy result to cpu
  cuda_pull_array(mm_out_gpu, mm_out, M*N);

  // cout << "Custom kernel output:" << endl;
  // print_3D_array(mm_out, w, h, num_ker);
  // print_sample_matrix(mm_out, M, N);

  cout << "cublasGemm \t\tMatrixMulRect" << endl;
  print_sample_side_by_side(gemm_out, mm_out, num_ker, w*h);

  //free memory
  free(im); free(weights); free(out); free(gemm_out); free(mm_out);
  cuda_free(im_gpu); cuda_free(weights_gpu); cuda_free(out_gpu); cuda_free(gemm_out_gpu); cuda_free(mm_out_gpu);

  // // Verify the result in CPU
  // verify_result(a, b, c, M, N, K);

  cout << "PROGRAM COMPLETELY SUCCESSFULLY!" << endl;

  return 0;
}
