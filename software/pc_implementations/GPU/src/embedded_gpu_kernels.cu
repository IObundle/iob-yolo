
#include <hip/hip_runtime.h>
/*
embedded_gpu_kernels.cu

implementations of GPU kernels for inference

*/

#ifdef GPU

#include "cuda_runtime.h"
#include "curand.h"
#include "cublas_v2.h"

extern "C"{
#include "embedded.h"
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

void fill_gpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(cudaPeekAtLastError());
}

__global__ void fill_fp_kernel(int N, int16_t ALPHA, int16_t *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

void fill_fp_gpu(int N, int16_t ALPHA, int16_t * X, int INCX)
{
    fill_fp_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(cudaPeekAtLastError());
}

#ifdef FIXED

__global__ void copy_fp2float_array_kernel(int N, int16_t *X, float *Y, int fracFP)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if(i < N) Y[i] = (float) ((X[i]*1.0)/((int32_t)(1<<fracFP)));
}

void copy_fp2float_array_gpu(int N, int16_t *X, float* Y, int fracFP)
{
  copy_fp2float_array_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, Y, fracFP);
  check_error(cudaPeekAtLastError());
}


__global__ void copy_float2fp_array_kernel(int N, float *X, int16_t *Y, int fracFP)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if(i < N) Y[i] = (int16_t) (X[i]*((int32_t)(1<<fracFP)));
}

void copy_float2fp_array_gpu(int N, float *X, int16_t* Y, int fracFP)
{
  copy_float2fp_array_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, Y, fracFP);
  check_error(cudaPeekAtLastError());
}

//im2col_gpu implementation
// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_fp_gpu_kernel(const int n, const int16_t* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        int16_t *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        int16_t* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const int16_t* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_fp_gpu(int16_t *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, int16_t *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_fp_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}


__global__ void add_bias_fp_kernel(int16_t *output, int16_t *biases, int batch, int n, int size, int bias_shift)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += (biases[j] >> bias_shift); //Q9.7 + Q9.7
}

void add_bias_fp_gpu(int16_t *output, int16_t *biases, int batch, int n, int size, int bias_shift)
{
    int num = n*size*batch;

    add_bias_fp_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size, bias_shift);
    check_error(cudaPeekAtLastError());
}


 
__global__ void add_bias_kernel(int16_t *output, int16_t *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void add_bias_gpu(int16_t *output, int16_t *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(cudaPeekAtLastError());
}

__device__ int16_t leaky_activate_kernel(int16_t x)
{
  //return (x>=0) ? x : (x >> 3); //Same as x*0.125
  return (x>=0) ? x : ((x >> 4) + (x >> 5) + (x >> 7)); //Same as x*0.1015625
}

__device__ int16_t logistic_activate_kernel(int16_t x)
{
  //linear sigmoid approximation
  // fp2375 = 2.375 in Q9.7, fp084375 = 0.84375 in Q9.7 ...
  int16_t fp2375 = 0x130, fp084375 = 0x6C, fp0625 = 0x50, fp05 = 0x40; //Q9.7
  int16_t fp5 = 0x280, fp1 = 0x080; //Q9.7
  int16_t val_in = x;

  if(val_in < 0) x = ~val_in +1; //emulates multiplying by -1
  
  if(x >= fp5) x = fp1; //saturated activation
  else if(x >= fp2375) x = fp084375 + (x >> 5); //0.84375 + x*2^(-5)
  else if(x >= fp1) x = fp0625 + (x >> 3); //0.625 + x*2^(-3)
  else x = fp05 + (x >> 2); //0.5 + x*2^(-2)

  return (val_in > 0)? (x) : (fp1 - x);
}

__global__ void activate_array_kernel(int16_t *x, int n, int a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    switch(a){
    case LEAKY:
      if(i < n) x[i] = leaky_activate_kernel(x[i]);
      break;
    case LOGISTIC:
      if(i < n) x[i] = logistic_activate_kernel(x[i]);
      break;
    default:
      break;
    }
}

void activate_array_gpu(int16_t *x, int n, int a) 
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
    check_error(cudaPeekAtLastError());
}

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, int16_t *input, int16_t *output, int *indexes)
{
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    int out_index = j + w*(i + h*(k + c*b));
    int16_t max = MIN_VAL_INT16;
    int max_i = -1;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            int16_t val = (valid != 0) ? input[index] : MIN_VAL_INT16;
            max_i = (val > max) ? index : max_i;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
    //indexes[out_index] = max_i;
}

/*
used arguments: w, h, num_ker, downsample, net_output_gpu, layer_num
unused arguments: ignorePadding, new_output_pos

*/
void forward_maxpool_layer_gpu(int w, int h, int num_ker, int downsample, int ignorePadding, unsigned int new_output_pos, void** net_output_gpu, int layer_num)
{
  /*
    stride = downsample + 1
    size = 2
    padding = size - 1 = 2 - 1 = 1
  */
    int out_h = (h+1-2)/(downsample+1) + 1; //(h + padding - size)/stride + 1 
    int out_w = (w+1-2)/(downsample+1) + 1; //(w + padding - size)/stride + 1 
    int c = num_ker;

    size_t n = out_h*out_w*c; //output size

    //create output array
    net_output_gpu[layer_num] = (void*) cuda_make_fp_array(0, n);
    
    forward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, h, w, num_ker, (downsample+1), 2, 1, (int16_t*) net_output_gpu[layer_num-1], (int16_t*) net_output_gpu[layer_num], 0);
    check_error(cudaPeekAtLastError());

}

__global__ void copy_fp_kernel(int N, int16_t *X, int OFFX, int INCX, int16_t *Y, int OFFY, int INCY)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

void copy_fp_gpu(int N, int16_t * X, int INCX,  int16_t * Y, int INCY)
{
  copy_fp_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, 0, INCX, Y, 0, INCY);
  check_error(cudaPeekAtLastError());

}



__global__ void upsample_kernel(size_t N, int16_t *x, int w, int h, int c, int batch, int stride, int forward, int scale, int16_t *out)
{
    size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int out_index = i;
    int out_w = i%(w*stride);
    i = i/(w*stride);
    int out_h = i%(h*stride);
    i = i/(h*stride);
    int out_c = i%c;
    i = i/c;
    int b = i%batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b*w*h*c + in_c*w*h + in_h*w + in_w;


    if(forward) out[out_index] += (int16_t) scale * x[in_index];
}

void upsample_gpu(int16_t *in, int w, int h, int c, int batch, int stride, int forward, int scale, int16_t *out)
{
    size_t size = w*h*c*batch*stride*stride;
    upsample_kernel<<<cuda_gridsize(size), BLOCK>>>(size, in, w, h, c, batch, stride, forward, scale, out);
    check_error(cudaPeekAtLastError());
}


//Shared version
#define SHMEM_SIZE (16*16)

/*
  Matrix multiplication kernel that leverages shared memory to reduce global GPU memory accesses
  Each input value is only accessed once.
  Each thread computes 1 output value.
  The output computation is divided into submatrices of size 16x16
*/
__global__ void gemm_fp_kernel(int16_t *a, int16_t *b, int16_t *c, int m, int n, int k){
	// shared memory
	__shared__ int A[SHMEM_SIZE];
	__shared__ int B[SHMEM_SIZE];

	// get global thread id
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// Extract some builtin values to simplify code
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dim = blockDim.x;

	//number of blocks along k dimension
	int num_k_blocks = (k + dim - 1)/dim;
	int idx_a, idx_b;
	
	// move the tile across the length of the grid
	int tmp = 0;
	// accumulate along k blocks
	for (int i = 0; i < num_k_blocks; i++){
	  // load values into shared memory
	  // Each thread in a block loads 1 value for each A and B
	  
	  // calculate matrix indexes
	  idx_a = (row*k) + (i*dim) + ty;
	  idx_b = (i*dim*n) + (tx*n) + col;
	  
	  // check input matrices boundaries
	  A[tx * dim + ty] = ( (row < m) && (i*dim+ ty < k)) ? (int) a[idx_a] : 0;
	  B[tx * dim + ty] = ( (i*dim+tx < k) && (col < n)) ? (int) b[idx_b] : 0;

	  __syncthreads();

	  //only perform muladd inside output boundaries
	  if(row < m && col < n){
	    //Accumulate the partial results
	    for (int j = 0; j < dim; j++){
	      tmp += A[tx*dim + j] * B[j*dim + ty]; //Q6.10 * Q9.7 = Q15.17
	    }
	  }
	  __syncthreads();
	}

	// write back result to main memory
	c[row*n + col] = (int16_t) (tmp >> 10); //Q15.17 -> Q9.7
}

// Naive Matrix Multiplication implementation for rectangular matrices
__global__ void matrixMulRect_kernel(int16_t *a, int16_t *b, int16_t *c, int M, int N, int K){
	// get thread global id
	int row  = blockIdx.x * blockDim.x + threadIdx.x;
	int col  = blockIdx.y * blockDim.y + threadIdx.y;

	// check range
	if(row < M && col < N){

		// Accumulate partial result
		int tmp = 0;

		for (int i = 0; i < K; i++){
		  tmp += ((int) a[row*K + i])* ((int) b[i*N + col]); // Q6.10*Q9.7 = Q15.17	
		}

		//Write back the result
		c[row*N + col] = (int16_t) (tmp>>10); //Q15.17 -> Q9.7
	}
}


void gemm_fp_gpu(int16_t *A_gpu, int16_t *B_gpu, int16_t *C_gpu, int M, int N, int K)
{
  //Create CTA and Grid dimentions
  int threads = 16; // 2D => 16 + 16 = 256
  int blocks_row = (M + threads - 1) / threads;
  int blocks_col = (N + threads - 1) / threads;

  // Setup our kernel launch parameters
  dim3 THREADS(threads, threads);
  dim3 BLOCKS(blocks_row, blocks_col);

  // naive matrix multiplication kernel - only accesses global memory
  matrixMulRect_kernel<<<BLOCKS, THREADS>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  // matrix multiplication kernel that uses shared memory
  // gemm_fp_kernel<<<BLOCKS, THREADS>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  check_error(cudaPeekAtLastError());
}



#else //Float
//im2col_gpu implementation
// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_gpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(cudaPeekAtLastError());
}

//__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .125f*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1015625f*x;}
__device__ float logistic_activate_kernel(float x){return 1.f/(1.f + expf(-x));}

__global__ void activate_array_kernel(float *x, int n, int a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    //tiny only used leaky activation, when batch_norm is done
    switch(a){
    case LEAKY:
      if(i < n) x[i] = leaky_activate_kernel(x[i]);
      break;
    case LOGISTIC:
      if(i < n) x[i] = logistic_activate_kernel(x[i]);
      break;
    default:
      break;
    }
}

void activate_array_gpu(float *x, int n, int a) 
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
    check_error(cudaPeekAtLastError());
}

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max_i = (val > max) ? index : max_i;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
    //indexes[out_index] = max_i;
}


/*
used arguments: w, h, num_ker, downsample, net_output_gpu, layer_num
unused arguments: ignorePadding, new_output_pos

*/
void forward_maxpool_layer_gpu(int w, int h, int num_ker, int downsample, int ignorePadding, unsigned int new_output_pos, void** net_output_gpu, int layer_num)
{
  /*
    stride = downsample + 1
    size = 2
    padding = size - 1 = 2 - 1 = 1
  */
    int out_h = (h+1-2)/(downsample+1) + 1; //(h + padding - size)/stride + 1 
    int out_w = (w+1-2)/(downsample+1) + 1; //(w + padding - size)/stride + 1 
    int c = num_ker;

    size_t n = out_h*out_w*c; //output size

    //create output array
    net_output_gpu[layer_num] = (void*) cuda_make_array(0, n);
    
    //    forward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.h, layer.w, layer.c, layer.stride, layer.size, layer.pad, net.input_gpu, layer.output_gpu, layer.indexes_gpu);
    // indexes is not used for anything?
    forward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, h, w, num_ker, (downsample+1), 2, 1, (float*) net_output_gpu[layer_num-1], (float*) net_output_gpu[layer_num], 0);
    check_error(cudaPeekAtLastError());

}

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int out_index = i;
    int out_w = i%(w*stride);
    i = i/(w*stride);
    int out_h = i%(h*stride);
    i = i/(h*stride);
    int out_c = i%c;
    i = i/c;
    int b = i%batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b*w*h*c + in_c*w*h + in_h*w + in_w;


    if(forward) out[out_index] += scale * x[in_index];
    else atomicAdd(x+in_index, scale * out[out_index]);
}

void upsample_gpu(float *in, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t size = w*h*c*batch*stride*stride;
    upsample_kernel<<<cuda_gridsize(size), BLOCK>>>(size, in, w, h, c, batch, stride, forward, scale, out);
    check_error(cudaPeekAtLastError());
}


#endif //ifdef FIXED

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}


void copy_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, 0, INCX, Y, 0, INCY);
    check_error(cudaPeekAtLastError());
}

#endif //ifdef GPU
